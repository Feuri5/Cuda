#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#define N 30
#define M 8 //Watki na blok

__global__ void add(int *a, int *b, int *c, int n) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index<n)
		c[index] = a[index] + b[index];
}

void random (int *tab, int wym )
{	
	int i;
	for(i=0;i<wym;i++)
		tab[i]=rand()%101;
}


int main(void) {

	clock_t cpu_time1, cpu_time2;

	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	
	int size = N * sizeof(int);
	srand(time(NULL));
	
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random(a, N);
	b = (int *)malloc(size); random(b, N);
	c = (int *)malloc(size);
	
	cpu_time1 = clock();
	for(int i = 0; i < N; i++)
	{
		c[i] = a[i] + b[i];
	
	}
	cpu_time2 = clock();
	double cpu_time = ((double) (cpu_time2 - cpu_time1)) / CLOCKS_PER_SEC;
	

	hipEvent_t c1, c2;
	hipEventCreate(&c1);
	hipEventCreate(&c2);
	
	hipEvent_t c3, c4;
	hipEventCreate(&c3);
	hipEventCreate(&c4);

	hipEventRecord(c3,0);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	hipEventRecord(c1,0);
	add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);
	hipEventRecord(c2,0);

	float gpu_time_without_copy = 0;

	cudaEventElaspedTime(&gpu_time_without_copy, c1, c2);


	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	hipEventRecord(c4,0);
	cudaEventSynchronise(c4);
	float gpu_time = 0;
	cudaEventElaspedTime(&gpu_time, c3, c4);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}


